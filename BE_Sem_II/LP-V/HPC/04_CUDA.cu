
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;
using namespace std::chrono;

// ---------------------- VECTOR ADDITION --------------------------
__global__ void vectorAddCUDA(int *a, int *b, int *c, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

void vectorAddCPU(int *a, int *b, int *c, int n)
{
    for (int i = 0; i < n; i++)
        c[i] = a[i] + b[i];
    // for (int i = 0; i < n; i++)
    //     cout << c[i] << " ";
    // cout << endl;
}

// ---------------------- MATRIX MULTIPLICATION ---------------------
__global__ void matrixMulCUDA(int *a, int *b, int *c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        int sum = 0;
        for (int k = 0; k < N; k++)
            sum += a[row * N + k] * b[k * N + col];
        c[row * N + col] = sum;
    }
}

void matrixMulCPU(int *a, int *b, int *c, int N)
{
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
        {
            int sum = 0;
            for (int k = 0; k < N; k++)
                sum += a[i * N + k] * b[k * N + j];
            c[i * N + j] = sum;
        }

    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         cout << c[i * N + j] << " ";
    //     }
    //     cout << endl;
    // }
}

// ------------------------ MAIN FUNCTION ---------------------------
int main()
{
    const int vecSize = 1 << 24; // ~16 million
    const int matrixSize = 1024; // 1024x1024 matrix

    // ------------------ Vector Addition ------------------
    int *h_a = new int[vecSize];
    int *h_b = new int[vecSize];
    int *h_c_cpu = new int[vecSize];
    int *h_c_gpu = new int[vecSize];

    for (int i = 0; i < vecSize; ++i)
    {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }

    auto start = high_resolution_clock::now();
    vectorAddCPU(h_a, h_b, h_c_cpu, vecSize);
    auto end = high_resolution_clock::now();
    cout << "[Vector Addition - CPU] Time: " << duration_cast<milliseconds>(end - start).count() << " ms\n";

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, vecSize * sizeof(int));
    hipMalloc(&d_b, vecSize * sizeof(int));
    hipMalloc(&d_c, vecSize * sizeof(int));

    hipMemcpy(d_a, h_a, vecSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, vecSize * sizeof(int), hipMemcpyHostToDevice);

    start = high_resolution_clock::now();
    vectorAddCUDA<<<(vecSize + 255) / 256, 256>>>(d_a, d_b, d_c, vecSize);
    hipDeviceSynchronize();
    end = high_resolution_clock::now();

    hipMemcpy(h_c_gpu, d_c, vecSize * sizeof(int), hipMemcpyDeviceToHost);
    cout << "[Vector Addition - GPU] Time: " << duration_cast<milliseconds>(end - start).count() << " ms\n\n";

    // ------------------ Matrix Multiplication ------------------
    int *matA = new int[matrixSize * matrixSize];
    int *matB = new int[matrixSize * matrixSize];
    int *matC_cpu = new int[matrixSize * matrixSize];
    int *matC_gpu = new int[matrixSize * matrixSize];

    for (int i = 0; i < matrixSize * matrixSize; ++i)
    {
        matA[i] = rand() % 100;
        matB[i] = rand() % 100;
    }

    start = high_resolution_clock::now();
    matrixMulCPU(matA, matB, matC_cpu, matrixSize);
    end = high_resolution_clock::now();
    cout << "[Matrix Multiplication - CPU] Time: " << duration_cast<milliseconds>(end - start).count() << " ms\n";

    int *d_matA, *d_matB, *d_matC;
    size_t bytes = matrixSize * matrixSize * sizeof(int);
    hipMalloc(&d_matA, bytes);
    hipMalloc(&d_matB, bytes);
    hipMalloc(&d_matC, bytes);

    hipMemcpy(d_matA, matA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((matrixSize + 15) / 16, (matrixSize + 15) / 16);

    start = high_resolution_clock::now();
    matrixMulCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_matA, d_matB, d_matC, matrixSize);
    hipDeviceSynchronize();
    end = high_resolution_clock::now();
    hipMemcpy(matC_gpu, d_matC, bytes, hipMemcpyDeviceToHost);

    cout << "[Matrix Multiplication - GPU] Time: " << duration_cast<milliseconds>(end - start).count() << " ms\n";

    // Cleanup
    delete[] h_a;
    delete[] h_b;
    delete[] h_c_cpu;
    delete[] h_c_gpu;
    delete[] matA;
    delete[] matB;
    delete[] matC_cpu;
    delete[] matC_gpu;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    return 0;
}